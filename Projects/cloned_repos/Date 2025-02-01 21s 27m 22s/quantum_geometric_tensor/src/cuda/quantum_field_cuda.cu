#include "hip/hip_runtime.h"
#include "quantum_geometric/physics/quantum_field_operations.h"
#include "quantum_geometric/hardware/quantum_geometric_cuda.h"
#include <hip/hip_runtime.h>

// CUDA parameters
#define BLOCK_SIZE 256
#define MAX_BLOCKS 65535

// CUDA kernels for field operations
__global__ void apply_rotation_kernel(
    QuantumAmplitude* field,
    size_t field_size,
    size_t num_components,
    size_t qubit,
    hipDoubleComplex* rotation) {
    
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= field_size) return;
    
    size_t mask = 1ULL << qubit;
    if (idx & mask) {
        // Get field components
        hipDoubleComplex psi_0 = to_cuda_complex(field[idx ^ mask].amplitude);
        hipDoubleComplex psi_1 = to_cuda_complex(field[idx].amplitude);
        
        // Apply rotation
        hipDoubleComplex new_psi_0 = hipCmul(rotation[0], psi_0);
        new_psi_0 = hipCadd(new_psi_0, hipCmul(rotation[1], psi_1));
        
        hipDoubleComplex new_psi_1 = hipCmul(rotation[2], psi_0);
        new_psi_1 = hipCadd(new_psi_1, hipCmul(rotation[3], psi_1));
        
        // Update field
        field[idx ^ mask].amplitude = from_cuda_complex(new_psi_0);
        field[idx].amplitude = from_cuda_complex(new_psi_1);
    }
}

__global__ void calculate_field_energy_kernel(
    QuantumAmplitude* field,
    QuantumAmplitude* momentum,
    size_t field_size,
    size_t num_components,
    double* energy) {
    
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= field_size) return;
    
    double local_energy = 0.0;
    
    // Kinetic energy
    for (size_t i = 0; i < num_components; i++) {
        hipDoubleComplex pi = to_cuda_complex(momentum[idx * num_components + i].amplitude);
        local_energy += hipCreal(hipCmul(pi, hipConj(pi)));
    }
    
    // Potential energy
    double phi_squared = 0.0;
    for (size_t i = 0; i < num_components; i++) {
        hipDoubleComplex phi = to_cuda_complex(field[idx * num_components + i].amplitude);
        phi_squared += hipCreal(hipCmul(phi, hipConj(phi)));
    }
    
    local_energy += 0.5 * phi_squared;
    
    // Atomic add to total energy
    atomicAdd(energy, local_energy);
}

// CUDA wrapper functions
extern "C" {

int apply_rotation_cuda(
    QuantumField* field,
    size_t qubit,
    double theta,
    double phi) {
    
    // Allocate device memory
    QuantumAmplitude* d_field;
    size_t field_size = field->field_tensor->size * sizeof(QuantumAmplitude);
    hipMalloc(&d_field, field_size);
    hipMemcpy(d_field, field->field_tensor->data, field_size, hipMemcpyHostToDevice);
    
    // Create rotation matrix
    hipDoubleComplex rotation[4];
    rotation[0] = make_hipDoubleComplex(cos(theta/2), 0);
    rotation[1] = make_hipDoubleComplex(-sin(theta/2) * cos(phi), -sin(theta/2) * sin(phi));
    rotation[2] = make_hipDoubleComplex(sin(theta/2) * cos(phi), sin(theta/2) * sin(phi));
    rotation[3] = make_hipDoubleComplex(cos(theta/2), 0);
    
    hipDoubleComplex* d_rotation;
    hipMalloc(&d_rotation, 4 * sizeof(hipDoubleComplex));
    hipMemcpy(d_rotation, rotation, 4 * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    
    // Launch kernel
    size_t num_blocks = (field->field_tensor->size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    if (num_blocks > MAX_BLOCKS) num_blocks = MAX_BLOCKS;
    
    apply_rotation_kernel<<<num_blocks, BLOCK_SIZE>>>(
        d_field,
        field->field_tensor->size,
        field->field_tensor->dims[4],
        qubit,
        d_rotation
    );
    
    // Copy result back
    hipMemcpy(field->field_tensor->data, d_field, field_size, hipMemcpyDeviceToHost);
    
    // Clean up
    hipFree(d_field);
    hipFree(d_rotation);
    
    return hipGetLastError() == hipSuccess ? 0 : -1;
}

double calculate_field_energy_cuda(const QuantumField* field) {
    // Allocate device memory
    QuantumAmplitude* d_field;
    QuantumAmplitude* d_momentum;
    double* d_energy;
    
    size_t field_size = field->field_tensor->size * sizeof(QuantumAmplitude);
    hipMalloc(&d_field, field_size);
    hipMalloc(&d_momentum, field_size);
    hipMalloc(&d_energy, sizeof(double));
    
    hipMemcpy(d_field, field->field_tensor->data, field_size, hipMemcpyHostToDevice);
    hipMemcpy(d_momentum, field->conjugate_momentum->data, field_size, hipMemcpyHostToDevice);
    hipMemset(d_energy, 0, sizeof(double));
    
    // Launch kernel
    size_t num_blocks = (field->field_tensor->size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    if (num_blocks > MAX_BLOCKS) num_blocks = MAX_BLOCKS;
    
    calculate_field_energy_kernel<<<num_blocks, BLOCK_SIZE>>>(
        d_field,
        d_momentum,
        field->field_tensor->size,
        field->field_tensor->dims[4],
        d_energy
    );
    
    // Get result
    double energy;
    hipMemcpy(&energy, d_energy, sizeof(double), hipMemcpyDeviceToHost);
    
    // Clean up
    hipFree(d_field);
    hipFree(d_momentum);
    hipFree(d_energy);
    
    return energy;
}

} // extern "C"
